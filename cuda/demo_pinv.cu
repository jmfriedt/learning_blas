
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <complex.h>
#include <hipblas.h>
#include <hipsolver.h>

#include <sys/time.h>

#undef debug
//#define debug

// https://stackoverflow.com/questions/15997888/creating-identity-matrix-with-cuda
__global__ void initIdentityGPU(hipDoubleComplex *devMatrix, int N) {
int x = blockDim.x*blockIdx.x + threadIdx.x;
if (x < N*N)
  if ((x/N) == (x%N)) {devMatrix[x].x = 1.; devMatrix[x].y = 0.;}
    else {devMatrix[x].x = 0.; devMatrix[x].x = 0.;}
}

// https://stackoverflow.com/questions/22887167/cublas-incorrect-inversion-for-matrix-with-zero-pivot
int main()
{ struct timeval tv1,tv2;
  int nobs=210000;
  int nlag=30;
  int l,m;
  const int N=2*nlag+1;
  hipDoubleComplex *dev_mem, *dev_mem_out, *dev_res, *dev_val, *dev_inv, *dev_Id, *dev_in;     // .x, .y
  std::complex<double> *host_mem,*host_res,*host_val,*host_code;  // real(), imag()
  hipblasHandle_t handle;
  hipDoubleComplex alpha,beta;
  alpha.x=1.;alpha.y=0.;
  beta.x=0;beta.y=0;
  
  hipSetDevice(0);
  hipblasCreate(&handle);

  hipMalloc((void **)&dev_mem, sizeof(hipDoubleComplex) * nobs * (nlag*2+1));
  hipMalloc((void **)&dev_mem_out, sizeof(hipDoubleComplex) * nobs * (nlag*2+1));
  host_mem=(std::complex<double>*)malloc(sizeof(std::complex<double>)*nobs*(nlag*2+1));
  host_val=(std::complex<double>*)malloc(sizeof(std::complex<double>)*nobs);
  host_code=(std::complex<double>*)malloc(sizeof(std::complex<double>)*nobs);
  host_res=(std::complex<double>*)malloc(sizeof(std::complex<double>)*(2*nlag+1)*(2*nlag+1));
  hipMalloc((void **)&dev_res, sizeof(hipDoubleComplex) * (2*nlag+1));
  hipMalloc((void **)&dev_val, sizeof(hipDoubleComplex) * nobs);
  hipMalloc((void **)&dev_inv, sizeof(hipDoubleComplex) * (2*nlag+1)*(2*nlag+1));
  hipMalloc((void **)&dev_in, sizeof(hipDoubleComplex) * (2*nlag+1)*(2*nlag+1));
  hipMalloc((void **)&dev_Id, sizeof(hipDoubleComplex) * (2*nlag+1)*(2*nlag+1));
  for (m=0;m<nobs;m++) 
      {host_val[m].real((double)(random()/pow(2,31))-0.5);
       host_val[m].imag((double)(random()/pow(2,31))-0.5);
       host_code[m].real((double)(random()/pow(2,31))-0.5);
       host_code[m].imag((double)(random()/pow(2,31))-0.5);
      }
//  for (m=0;m<nlag;m++) printf("%lf ",real(host_val[m]));
//  printf("\n");
  for (m=0;m<nobs-12;m++)          // time shifted copies of the code
      {host_val[m+10]+=host_code[m]*.3;
       host_val[m+5]+=host_code[m]*.5;
       host_val[m]+=host_code[m+12]*.7;
       host_val[m]+=host_code[m+3];
      }
  memset(host_mem , 0x0, sizeof(std::complex<double>) * nobs * (2*nlag+1));
  for (l=-nlag;l<=nlag;l++)
    for (m=0;m<nobs-(l+nlag);m++)
       if (l<0) host_mem[(m)+nobs*(l+nlag)]=host_code[m-l];
          else  host_mem[(m+l)+nobs*(l+nlag)]=host_code[m];
      // host_mem[(l+nlag)*nobs+m+l+nlag]=host_code[m];
  hipblasSetMatrix (nobs, nlag*2+1, sizeof(*host_mem), host_mem, nobs, dev_mem, nobs);
  hipblasSetMatrix (1, nobs, sizeof(*host_val), host_val, 1   , dev_val, 1   );
  int *P, *INFO;
  hipMalloc((void **)&P, sizeof(int) * (2*nlag+1));
  hipMalloc((void **)&INFO, sizeof(int));
  hipsolverHandle_t handlegetrs = NULL;
  int bufferSize = 0;
  hipDoubleComplex *buffer = NULL;
  initIdentityGPU<<<128, 128>>>(dev_Id,N); // fill Identity matrix
/*
  for (m=0;m<(2*nlag+1);m++)
    for (l=0;l<(2*nlag+1);l++) 
      if (m!=l) {host_res[m+l*(2*nlag+1)].real(0.);host_res[m+l*(2*nlag+1)].imag(0.);}
      else {host_res[m+l*(2*nlag+1)].real(1.);host_res[m+l*(2*nlag+1)].imag(0.);}
  cudaMemcpy(dev_Id,host_res,sizeof(cuDoubleComplex) * (2*nlag+1)*(2*nlag+1),cudaMemcpyHostToDevice);
*/
#ifdef debug
  cudaDeviceSynchronize();
  cudaMemcpy(host_res,dev_Id,sizeof(cuDoubleComplex) * (2*nlag+1)*(2*nlag+1),cudaMemcpyDeviceToHost);
  printf("Id\n");
  for (m=0;m<(2*nlag+1);m++)
    {for (l=0;l<(2*nlag+1);l++) printf("%.4lf ",real(host_res[m+l*(2*nlag+1)]));
     printf("; \n");
    }
#endif
  gettimeofday(&tv1,NULL);
  if (hipblasZgemm(handle, HIPBLAS_OP_C, HIPBLAS_OP_N, N, N, nobs, &alpha, dev_mem,  nobs, dev_mem, nobs, &beta, dev_in, N) != HIPBLAS_STATUS_SUCCESS)
     printf("error 0\n");
#ifdef debug
  int INFOh;
  memset(host_res , 0x0, sizeof(std::complex<double>) * (2*nlag+1) * (2*nlag+1));
  cudaMemcpy(host_res,dev_in,sizeof(cuDoubleComplex) * (2*nlag+1)*(2*nlag+1),cudaMemcpyDeviceToHost);
  printf("Id\n");
  for (m=0;m<(2*nlag+1);m++)
    {for (l=0;l<(2*nlag+1);l++) printf("%.0lf ",real(host_res[m+l*(2*nlag+1)]));
     printf("; \n");
    }
#endif
  hipsolverDnCreate(&handlegetrs);
  hipsolverDnZgetrf_bufferSize(handlegetrs, N, N, dev_in, N, &bufferSize);
  hipMalloc(&buffer, sizeof(hipDoubleComplex) * bufferSize );
//  cudaMalloc(&buffer, sizeof(cuDoubleComplex) * N );
// https://docs.nvidia.com/cuda/cusolver/index.html
  if (hipsolverDnZgetrf(handlegetrs, N, N, dev_in, N, buffer, P, INFO) != HIPSOLVER_STATUS_SUCCESS)
     printf("error 1\n");
#ifdef debug
  cudaDeviceSynchronize();
  cudaMemcpy(&INFOh,INFO,sizeof(int),cudaMemcpyDeviceToHost);
  printf("INFO: %d\n",INFOh);
#endif
  if (hipsolverDnZgetrs(handlegetrs, HIPBLAS_OP_N, N, N, dev_in, N, P, dev_Id, N, INFO) != HIPSOLVER_STATUS_SUCCESS)
     printf("error 2\n");
#ifdef debug
  cudaMemcpy(&INFOh,INFO,sizeof(int),cudaMemcpyDeviceToHost);
  printf("INFO: %d\n",INFOh);
  cudaMemcpy(host_res,dev_Id,sizeof(cuDoubleComplex) * (2*nlag+1)*(2*nlag+1),cudaMemcpyDeviceToHost);
  printf("res\n");
  for (m=0;m<(2*nlag+1);m++)
    {for (l=0;l<(2*nlag+1);l++) printf("%.4lf ",real(host_res[m+l*(2*nlag+1)]));
     printf("; \n");
    }
#endif
  if (hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, nobs, 2*nlag+1, 2*nlag+1, &alpha, dev_mem,  nobs, dev_Id, 2*nlag+1, &beta, dev_mem_out, nobs) != HIPBLAS_STATUS_SUCCESS)
     printf("error 3\n");
  // /!\ output matrix must NOT be the same than input argument ("in-place computation is not allowed", "C must not overlap")
#ifdef debug
  cudaMemcpy(host_mem,dev_mem_out,sizeof(cuDoubleComplex) * (2*nlag+1) * (nobs),cudaMemcpyDeviceToHost);
  printf("res\n");
  for (m=0;m<nobs;m++)
    {for (l=0;l<(2*nlag+1);l++) printf("%.4lf ",real(host_mem[m+l*(nobs)]));
     printf("; \n");
    }
#endif
  hipblasZgemm(handle, HIPBLAS_OP_C, HIPBLAS_OP_N, 1, 2*nlag+1, nobs, &alpha, dev_val,  nobs, dev_mem_out, nobs, &beta, dev_res, 1);
  hipDeviceSynchronize();
  gettimeofday(&tv2,NULL);
  printf("\ntime %ld\n",(tv2.tv_sec-tv1.tv_sec)*1000000+tv2.tv_usec-tv1.tv_usec);
  hipMemcpy(host_res,dev_res,sizeof(hipDoubleComplex) * (2*nlag+1),hipMemcpyDeviceToHost);
  for (m=0;m<2*nlag+1;m++) printf("%.9lf ",abs(host_res[m]));
  printf("\n");
  hipFree(P), hipFree(INFO), hipblasDestroy(handle);
}

/*
#include <lapacke.h>
  cblas_zgemm(CblasColMajor, CblasConjTrans, CblasNoTrans, 2*nlag+1, 2*nlag+1, nobs, &alpha, host_mem, nobs, host_mem, nobs, &beta, host_res, 2*nlag+1 ); 
  zgetrf_(&N,&N,reinterpret_cast <__complex__ double*>(host_res),&N,IPIV,&info); // LU decomposition: modifies input to output
  zgetri_(&N,reinterpret_cast <__complex__ double*>(host_res),&N,IPIV,reinterpret_cast <__complex__ double*>(WORK),&LWORK,&info); // inverse
  cblas_zgemm(CblasColMajor, CblasNoTrans, CblasNoTrans, nobs, 2*nlag+1, 2*nlag+1, &alpha, host_mem, nobs, host_res, 2*nlag+1, &beta, host_out, nobs ); 
  cblas_zgemm(CblasColMajor, CblasConjTrans, CblasNoTrans, 1, 2*nlag+1, nobs, &alpha, host_val, nobs, host_out, nobs, &beta, host_final, 1 ); */
