
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <complex.h>
#include <hipblas.h>
#include <hipsolver.h>

#include <sys/time.h>

#define PI 3.141592653589793

#undef debug

// https://stackoverflow.com/questions/15997888/creating-identity-matrix-with-cuda
__global__ void initIdentityGPU(hipDoubleComplex *devMatrix, int N) {
int x = blockDim.x*blockIdx.x + threadIdx.x;
int y = blockDim.y*blockIdx.y + threadIdx.y;
if(y < N && x < N) {
  if(x == y) {devMatrix[y*N+x].x = 1.; devMatrix[y*N+x].y = 0.;}
    else {devMatrix[y*N+x].x = 0.; devMatrix[y*N+x].x = 0.;}
  }
}

// https://stackoverflow.com/questions/22887167/cublas-incorrect-inversion-for-matrix-with-zero-pivot
int main()
{
  int nobs=2100;
  int nlag=15; // MUST BE < 16 !
  int l,m;
  const int N=2*nlag+1;
  hipDoubleComplex *dev_mem, *dev_res, *dev_val, *dev_inv, *dev_Id, *dev_in;     // .x, .y
  std::complex<double> *host_mem,*host_res,*host_val,*host_code;  // real(), imag()
  hipblasHandle_t handle;
  hipDoubleComplex alpha,beta;
  alpha.x=1.;alpha.y=0.;
  beta.x=0;beta.y=0;
  
  hipSetDevice(0);
  hipblasCreate(&handle);

  hipMalloc((void **)&dev_mem, sizeof(hipDoubleComplex) * nobs * (nlag*2+1));
  host_mem=(std::complex<double>*)malloc(sizeof(std::complex<double>)*nobs*(nlag*2+1));
  host_val=(std::complex<double>*)malloc(sizeof(std::complex<double>)*nobs);
  host_code=(std::complex<double>*)malloc(sizeof(std::complex<double>)*nobs);
  host_res=(std::complex<double>*)malloc(sizeof(std::complex<double>)*(2*nlag+1)*(2*nlag+1));
  hipMalloc((void **)&dev_res, sizeof(hipDoubleComplex) * (2*nlag+1));
  hipMalloc((void **)&dev_val, sizeof(hipDoubleComplex) * nobs);
  hipMalloc((void **)&dev_inv, sizeof(hipDoubleComplex) * (2*nlag+1)*(2*nlag+1));
  hipMalloc((void **)&dev_in, sizeof(hipDoubleComplex) * (2*nlag+1)*(2*nlag+1));
  hipMalloc((void **)&dev_Id, sizeof(hipDoubleComplex) * (2*nlag+1)*(2*nlag+1));
  for (m=0;m<nobs;m++) 
      {host_val[m].real((double)(random()/pow(2,31))-0.5);
       host_val[m].imag((double)(random()/pow(2,31))-0.5);
       host_code[m].real((double)(random()/pow(2,31))-0.5);
       host_code[m].imag((double)(random()/pow(2,31))-0.5);
      }
//  for (m=0;m<nlag;m++) printf("%lf ",real(host_val[m]));
//  printf("\n");
  for (m=0;m<nobs-12;m++)          // time shifted copies of the code
      {host_val[m+10]+=host_code[m]*.3;
       host_val[m+5]+=host_code[m]*.5;
       host_val[m]+=host_code[m+12]*.7;
       host_val[m]+=host_code[m+3];
      }

  memset(host_mem , 0x0, sizeof(std::complex<double>) * nobs * (2*nlag+1));
  for (l=-nlag;l<=nlag;l++)
    for (m=0;m<nobs-(l+nlag);m++)
       if (l<0) host_mem[(m)+nobs*(l+nlag)]=host_code[m-l];
          else  host_mem[(m+l)+nobs*(l+nlag)]=host_code[m];
      // host_mem[(l+nlag)*nobs+m+l+nlag]=host_code[m];
  hipblasSetMatrix (nobs, nlag*2+1, sizeof(*host_mem), host_mem, nobs, dev_mem, nobs);
  hipblasSetMatrix (1, nobs, sizeof(*host_val), host_val, 1   , dev_val, 1   );
  if (hipblasZgemm(handle, HIPBLAS_OP_C, HIPBLAS_OP_N, N, N, nobs, &alpha, dev_mem,  nobs, dev_mem, nobs, &beta, dev_in, N) != HIPBLAS_STATUS_SUCCESS)
//	 printf("error 0\n");
  hipDeviceSynchronize();
  int *P, *INFO;
  hipMalloc((void **)&P, sizeof(int) * (2*nlag+1));
  hipMalloc((void **)&INFO, sizeof(int));
//  (cudaMalloc<int>(&P,N * sizeof(int)));
//  (cudaMalloc<int>(&INFO,sizeof(int)));
/*
  cublasZgetrfBatched(handle, N, A_d, N, P, INFO,1)
  cublasZgetriBatched(handle, N, A_d, N, P, C_d, N , INFO, 1)
*/

  hipsolverHandle_t handlegetrs = NULL;
  int bufferSize = 0;
  hipDoubleComplex *buffer = NULL;
//  initIdentityGPU<<<128, 128>>>(dev_Id,N);
  for (m=0;m<(2*nlag+1);m++)
    for (l=0;l<(2*nlag+1);l++) 
      if (m!=l) {host_res[m+l*(2*nlag+1)].real(0.);host_res[m+l*(2*nlag+1)].imag(0.);}
      else {host_res[m+l*(2*nlag+1)].real(1.);host_res[m+l*(2*nlag+1)].imag(0.);}
  hipMemcpy(dev_Id,host_res,sizeof(hipDoubleComplex) * (2*nlag+1)*(2*nlag+1),hipMemcpyHostToDevice);
  memset(host_res , 0x0, sizeof(std::complex<double>) * (2*nlag+1) * (2*nlag+1));
#ifdef debug
  cudaMemcpy(host_res,dev_in,sizeof(cuDoubleComplex) * (2*nlag+1)*(2*nlag+1),cudaMemcpyDeviceToHost);
  printf("Id\n");
  for (m=0;m<(2*nlag+1);m++)
    {for (l=0;l<(2*nlag+1);l++) printf("%.0lf ",real(host_res[m+l*(2*nlag+1)]));
     printf("; \n");
    }
#endif
  hipsolverDnCreate(&handlegetrs);
  hipsolverDnZgetrf_bufferSize(handlegetrs, N, N, dev_in, N, &bufferSize);
  hipMalloc(&buffer, sizeof(hipDoubleComplex) * bufferSize );
// https://docs.nvidia.com/cuda/cusolver/index.html
  if (hipsolverDnZgetrf(handlegetrs, N, N, dev_in, N, buffer, P, INFO) != HIPSOLVER_STATUS_SUCCESS)
     printf("error 1\n");
  if (hipsolverDnZgetrs(handlegetrs, HIPBLAS_OP_N, N, N, dev_in, N, P, dev_Id, N, INFO) != HIPSOLVER_STATUS_SUCCESS)
     printf("error 2\n");
  hipMemcpy(host_res,dev_Id,sizeof(hipDoubleComplex) * (2*nlag+1)*(2*nlag+1),hipMemcpyDeviceToHost);
#ifdef debug
  printf("res\n");
  for (m=0;m<(2*nlag+1);m++)
    {for (l=0;l<(2*nlag+1);l++) printf("%.4lf ",real(host_res[m+l*(2*nlag+1)]));
     printf("; \n");
    }
#endif
  if (hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, nobs, 2*nlag+1, 2*nlag+1, &alpha, dev_mem,  nobs, dev_Id, 2*nlag+1, &beta, dev_mem, nobs) != HIPBLAS_STATUS_SUCCESS)
     printf("error 3\n");
  hipblasZgemm(handle, HIPBLAS_OP_C, HIPBLAS_OP_N, 1, 2*nlag+1, nobs, &alpha, dev_val,  nobs, dev_mem, nobs, &beta, dev_res, 1);
  hipMemcpy(host_res,dev_res,sizeof(hipDoubleComplex) * (2*nlag+1),hipMemcpyDeviceToHost);
  for (m=0;m<2*nlag+1;m++) printf("%.2lf ",abs(host_res[m]));
  printf("\n");
  hipFree(P), hipFree(INFO), hipblasDestroy(handle);
}

/*
#include <lapacke.h>
  cblas_zgemm(CblasColMajor, CblasConjTrans, CblasNoTrans, 2*nlag+1, 2*nlag+1, nobs, &alpha, host_mem, nobs, host_mem, nobs, &beta, host_res, 2*nlag+1 ); 
  zgetrf_(&N,&N,reinterpret_cast <__complex__ double*>(host_res),&N,IPIV,&info); // LU decomposition: modifies input to output
  zgetri_(&N,reinterpret_cast <__complex__ double*>(host_res),&N,IPIV,reinterpret_cast <__complex__ double*>(WORK),&LWORK,&info); // inverse
  cblas_zgemm(CblasColMajor, CblasNoTrans, CblasNoTrans, nobs, 2*nlag+1, 2*nlag+1, &alpha, host_mem, nobs, host_res, 2*nlag+1, &beta, host_out, nobs ); 
  cblas_zgemm(CblasColMajor, CblasConjTrans, CblasNoTrans, 1, 2*nlag+1, nobs, &alpha, host_val, nobs, host_out, nobs, &beta, host_final, 1 ); */
