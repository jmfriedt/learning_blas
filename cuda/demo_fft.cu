#include "hip/hip_runtime.h"
// FFT et iFFT

// nvcc demo_fft.cu -o demo_fft -lcufft -lm
#include <stdio.h>
#include <math.h>
#include <complex.h>
#include <hipfft/hipfft.h>

#include <fftw3.h> // nvcc demo_fft.cu -o demo_fft -lcufft -lm -lfftw3

#include <sys/time.h>

#define PI 3.141592653589793

int main()
{
  //hipblasHandle_t handle;
  int nobs=10000000;
  int k;
  double f=440.,fs=48000.,ph=0.;
  hipfftDoubleComplex *dev_mem;     // .x, .y
  std::complex<double> *host_mem;  // real(), imag()
  struct timeval tv1,tv2;

  hipfftHandle plan;
  hipSetDevice(0);
  //hipblasCreate(&handle);
  hipMalloc((void **)&dev_mem, sizeof(hipfftDoubleComplex) * nobs);
  host_mem=(std::complex<double>*)malloc(sizeof(std::complex<double>)*nobs);
  for (k=0;k<nobs;k++)
  {host_mem[k].real(cos(ph));
   host_mem[k].imag(sin(ph));
   ph+=2*PI*f/fs;if (ph>2*PI) ph-=2*PI;
   if (k<20) printf("%.2lf ",real(host_mem[k]));
  }
  printf("\n");
  hipMemcpy(dev_mem, host_mem, sizeof(hipfftDoubleComplex) * nobs, hipMemcpyHostToDevice);
  gettimeofday(&tv1,NULL);
  hipfftPlan1d(&plan, nobs, HIPFFT_Z2Z, 1);
  gettimeofday(&tv2,NULL); printf("\ntime %d\n",tv2.tv_usec-tv1.tv_usec);
  gettimeofday(&tv1,NULL);
  hipfftExecZ2Z(plan, dev_mem, dev_mem, HIPFFT_FORWARD);
  gettimeofday(&tv2,NULL); printf("\ntime %d\n",tv2.tv_usec-tv1.tv_usec);
  hipMemcpy(host_mem, dev_mem, sizeof(hipfftDoubleComplex) * nobs, hipMemcpyDeviceToHost);
  for (k=0;k<20;k++) printf("%.2lf ",abs(host_mem[k+(int)(f/fs*(float)nobs)-10])); // 440/fs*nobs
  gettimeofday(&tv1,NULL);
  hipfftExecZ2Z(plan, dev_mem, dev_mem, HIPFFT_BACKWARD);
  gettimeofday(&tv2,NULL); printf("\ntime %d\n",tv2.tv_usec-tv1.tv_usec);
  hipfftDestroy(plan);
  
  printf("\n");
  hipMemcpy(host_mem, dev_mem, sizeof(hipfftDoubleComplex) * nobs, hipMemcpyDeviceToHost);
  for (k=0;k<20;k++) printf("%.2lf ",real(host_mem[k])/(double)nobs);
  printf("\n");

  ph=0.;
  for (k=0;k<nobs;k++)
  {host_mem[k].real(cos(ph));
   host_mem[k].imag(sin(ph));
   ph+=2*PI*f/fs;if (ph>2*PI) ph-=2*PI;
   if (k<20) printf("%.2lf ",real(host_mem[k]));
  }
  printf("\n");
  fftw_plan _plan_a_dx;
  fftw_plan _ifft_dx;
  _plan_a_dx = fftw_plan_dft_1d(nobs,
     reinterpret_cast<fftw_complex*>(host_mem), reinterpret_cast<fftw_complex*>(host_mem),
     FFTW_FORWARD, FFTW_ESTIMATE);
  _ifft_dx = fftw_plan_dft_1d(nobs,
     reinterpret_cast<fftw_complex*>(host_mem), reinterpret_cast<fftw_complex*>(host_mem),
     FFTW_BACKWARD, FFTW_ESTIMATE);
  gettimeofday(&tv1,NULL);
  fftw_execute(_plan_a_dx);
  gettimeofday(&tv2,NULL); printf("\ntime %d\n",tv2.tv_usec-tv1.tv_usec);
  for (k=0;k<20;k++) printf("%.2lf ",abs(host_mem[k+(int)(f/fs*(float)nobs)-10])); // 440/fs*nobs
  gettimeofday(&tv1,NULL);
  fftw_execute(_ifft_dx);
  gettimeofday(&tv2,NULL); printf("\ntime %d\n",tv2.tv_usec-tv1.tv_usec);
  for (k=0;k<20;k++) printf("%.2lf ",real(host_mem[k])/(double)nobs);
}
