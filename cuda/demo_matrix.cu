#include "hip/hip_runtime.h"
//nvcc demo_matrix.cu -o demo_matrix -lcublas -lm
#include <stdio.h>
#include <math.h>
#include <complex.h>
#include <hipblas.h>

#include <gsl/gsl_fit.h>

#include <sys/time.h>

#define PI 3.141592653589793

int main()
{
  //hipblasHandle_t handle;
  int nobs=2100;
  int nlag=20;
  int l,m;
  struct timeval tv1,tv2;
  hipDoubleComplex *dev_mem, *dev_res, *dev_val;     // .x, .y
  std::complex<double> *host_mem,*host_res,*host_val,*host_code;  // real(), imag()
  hipblasHandle_t handle;
  hipDoubleComplex pwr;
  hipDoubleComplex alpha,beta;
  alpha.x=1.;alpha.y=0.;
  beta.x=0;beta.y=0;

  hipSetDevice(0);
  hipblasCreate(&handle);
  hipMalloc((void **)&dev_mem, sizeof(hipDoubleComplex) * nobs * (nlag*2+1));
  host_mem=(std::complex<double>*)malloc(sizeof(std::complex<double>)*nobs*(nlag*2+1));
  for (m=0;m<nobs;m++)
    {host_mem[m].real((double((m)%8)));
     host_mem[m].imag((double((m)%9)));
    }
  hipMemcpy(dev_mem, host_mem, sizeof(hipDoubleComplex) * nobs , hipMemcpyHostToDevice);
  gettimeofday(&tv1,NULL);
  hipblasZdotc(handle, nobs, dev_mem, 1, dev_mem, 1, &pwr);
  gettimeofday(&tv2,NULL); printf("\ntime %d\n",tv2.tv_usec-tv1.tv_usec);
  printf("power %lf\n",sqrt(pwr.x*pwr.x+pwr.y+pwr.y));

/*
octave:9> a=mod([0:99],8)+j*mod([0:99],9);
octave:10> (a*a')^2
3938
aussi sum(abs(a).^2)^2
power 3938.000000
*/

//https://www.netlib.org/lapack/explore-html/d1/d54/group__double__blas__level3_gaeda3cbd99c8fb834a60a6412878226e1.html
  host_val=(std::complex<double>*)malloc(sizeof(std::complex<double>)*nobs);
  host_code=(std::complex<double>*)malloc(sizeof(std::complex<double>)*nobs);
  host_res=(std::complex<double>*)malloc(sizeof(std::complex<double>)*(2*nlag+1));
  hipMalloc((void **)&dev_res, sizeof(hipDoubleComplex) * (2*nlag+1));
  hipMalloc((void **)&dev_val, sizeof(hipDoubleComplex) * nobs);
  for (m=0;m<nobs;m++) 
      {host_val[m].real((double)(random()/pow(2,31))-0.5);
       host_val[m].imag((double)(random()/pow(2,31))-0.5);
       host_code[m].real((double)(random()/pow(2,31))-0.5);
       host_code[m].imag((double)(random()/pow(2,31))-0.5);
      }
//  for (m=0;m<nlag;m++) printf("%lf ",real(host_val[m]));
//  printf("\n");
  for (m=0;m<nobs-12;m++)          // time shifted copies of the code
      {host_val[m+10]+=host_code[m];
       host_val[m+5]+=host_code[m];
       host_val[m]+=host_code[m+12];
       host_val[m]+=host_code[m+3];
      }

  memset(host_mem , 0x0, sizeof(std::complex<double>) * nobs * nlag);
  for (l=-nlag;l<=nlag;l++)
    for (m=0;m<nobs-(l+nlag);m++)
       if (l<0) host_mem[(m)+nobs*(l+nlag)]=host_code[m-l];
          else  host_mem[(m+l)+nobs*(l+nlag)]=host_code[m];
      // host_mem[(l+nlag)*nobs+m+l+nlag]=host_code[m];
  hipblasSetMatrix (nobs, nlag*2+1, sizeof(*host_mem), host_mem, nobs, dev_mem, nobs);
  hipblasSetMatrix (1, nobs, sizeof(*host_val), host_val, 1   , dev_val, 1   );
//  hipMemcpy(dev_mem, host_mem, sizeof(hipDoubleComplex) * nobs * nlag, hipMemcpyHostToDevice);
//  hipMemcpy(dev_val, host_val, sizeof(hipDoubleComplex) * nobs       , hipMemcpyHostToDevice);
//correlation et position du max
  hipblasZgemm(handle, HIPBLAS_OP_C, HIPBLAS_OP_N, 1, 2*nlag+1, nobs, &alpha, dev_val,  nobs, dev_mem, nobs, &beta, dev_res, 1);
//   ** On entry to ZGEMM  parameter number 8 had an illegal value si 1 au lieu de nobs apres dev_val
//  hipMemcpy(host_res, dev_res, sizeof(hipDoubleComplex) * nlag       , hipMemcpyDeviceToHost);
  hipblasGetMatrix (1, 2*nlag+1, sizeof(*host_res), dev_res, 1, host_res, 1);
// hipblasIdamax(handle, ci[i].nlag * 2 + 1, ci[i].dev_cor + p * (ci[i].nlag * 2 + 1), 1, &pk_idx);
// pk_idx -= 1;
  for (m=0;m<2*nlag+1;m++) printf("%.2lf ",abs(host_res[m]));
  printf("\n");
}
