#include "hip/hip_runtime.h"
//nvcc demo_matrix.cu -o demo_matrix -lcublas -lm
#include <stdio.h>
#include <math.h>
#include <complex.h>
#include <hipblas.h>

#include <gsl/gsl_fit.h>

#include <sys/time.h>

#define PI 3.141592653589793

int main()
{
  //hipblasHandle_t handle;
  int nobs=100;
  int nlag=20;
  int l,m;
  struct timeval tv1,tv2;
  hipDoubleComplex *dev_mem, *dev_res, *dev_val;     // .x, .y
  std::complex<double> *host_mem,*host_res,*host_val,*host_code;  // real(), imag()
  hipblasHandle_t handle;
  hipDoubleComplex pwr;
  hipDoubleComplex alpha,beta;
  alpha.x=1.;alpha.y=0.;
  beta.x=0;beta.y=0;

  hipSetDevice(0);
  hipblasCreate(&handle);
  hipMalloc((void **)&dev_mem, sizeof(hipDoubleComplex) * nobs * nlag*2);
  host_mem=(std::complex<double>*)malloc(sizeof(std::complex<double>)*nobs*nlag*2);
  for (m=0;m<nobs;m++)
    {host_mem[m].real((double((m)%8)));
     host_mem[m].imag((double((m)%9)));
    }
  hipMemcpy(dev_mem, host_mem, sizeof(hipDoubleComplex) * nobs , hipMemcpyHostToDevice);
  gettimeofday(&tv1,NULL);
  hipblasZdotc(handle, nobs, dev_mem, 1, dev_mem, 1, &pwr);
  gettimeofday(&tv2,NULL); printf("\ntime %d\n",tv2.tv_usec-tv1.tv_usec);
  printf("power %lf\n",sqrt(pwr.x*pwr.x+pwr.y+pwr.y));

/*
octave:9> a=mod([0:99],8)+j*mod([0:99],9);
octave:10> (a*a')^2
3938
aussi sum(abs(a).^2)^2
power 3938.000000
*/

//https://www.netlib.org/lapack/explore-html/d1/d54/group__double__blas__level3_gaeda3cbd99c8fb834a60a6412878226e1.html
  host_val=(std::complex<double>*)malloc(sizeof(std::complex<double>)*nobs);
  host_code=(std::complex<double>*)malloc(sizeof(std::complex<double>)*nobs);
  host_res=(std::complex<double>*)malloc(sizeof(std::complex<double>)*nlag);
  hipMalloc((void **)&dev_res, sizeof(hipDoubleComplex) * nlag);
  hipMalloc((void **)&dev_val, sizeof(hipDoubleComplex) * nobs);
  for (m=0;m<nobs;m++) 
      {host_val[m].real((double)(random()/pow(2,31))-0.5);
       host_val[m].imag((double)(random()/pow(2,31))-0.5);
       host_code[m].real((double)(random()/pow(2,31))-0.5);
       host_code[m].imag((double)(random()/pow(2,31))-0.5);
      }
  for (m=0;m<nlag;m++) printf("%lf ",real(host_val[m]));
  printf("\n");
  for (m=0;m<nobs-12;m++)          // time shifted copies of the code
      {host_val[m+12]+=host_code[m];
       host_val[m+3]+=host_code[m];
       host_val[m]+=host_code[m+12];
       host_val[m]+=host_code[m+3];
      }

  memset(host_mem , 0x0, sizeof(std::complex<double>) * nobs * nlag);
  for (l=-nlag;l<nlag;l++)
    for (m=0;m<nobs-l;m++)
      host_mem[(l+nlag)*nobs+m+l+nlag]=host_code[m];
  hipblasSetMatrix (nobs, nlag*2, sizeof(*host_mem), host_mem, nobs, dev_mem, nobs);
  hipblasSetMatrix (1, nobs, sizeof(*host_val), host_val, 1   , dev_val, 1   );
//  hipMemcpy(dev_mem, host_mem, sizeof(hipDoubleComplex) * nobs * nlag, hipMemcpyHostToDevice);
//  hipMemcpy(dev_val, host_val, sizeof(hipDoubleComplex) * nobs       , hipMemcpyHostToDevice);
//correlation et position du max
  //hipblasZgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 1, nlag, nobs, &alpha, dev_val, nobs, dev_mem, nobs, &beta, dev_res, 1);
  //                  transpose    no transpose m   n      k     1.     mxk       k    kxn     k      0.     res    m
  hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 1, nlag, nobs, &alpha, dev_val,   1 , dev_mem, nobs, &beta, dev_res, 1);
  //                  transpose    no transpose m   n      k     1.     mxk       m    kxn     k      0.     res    m
  //hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, 1, nlag, nobs, &alpha, dev_val,   1 , dev_mem, nlag, &beta, dev_res, 1);
  //                  transpose    no transpose m   n      k     1.     mxk       m    kxn     n      0.     res    m
  //hipblasZgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, 1, nlag, nobs, &alpha, dev_val, nobs, dev_mem, nlag, &beta, dev_res, 1);
  //                  transpose    no transpose m   n      k     1.     mxk       k    kxn     n      0.     res    m
     // C := alpha*op( A )*op( B ) + beta*C,
     // alpha and beta are scalars, and A, B and C are matrices, with op( A )
     // an m by k matrix,  op( B )  a  k by n matrix and  C an m by n matrix.
     // (T/N,T/N,m,n,k,alpha, A,m/k selon N ou T, B, k/n selon N ou T, beta, C, m)
// ** On entry to ZGEMM  parameter number 8 had an illegal value
//  hipMemcpy(host_res, dev_res, sizeof(hipDoubleComplex) * nlag       , hipMemcpyDeviceToHost);
  hipblasGetMatrix (1, nlag, sizeof(*host_res), dev_res, 1, host_res, 1);
  // hipblasIdamax(handle, ci[i].nlag * 2 + 1, ci[i].dev_cor + p * (ci[i].nlag * 2 + 1), 1, &pk_idx);
  // pk_idx -= 1;
  for (m=0;m<nlag;m++) printf("%lf ",abs(host_res[m]));
  printf("\n");
}


